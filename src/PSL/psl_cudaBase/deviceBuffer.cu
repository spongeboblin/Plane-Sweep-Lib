#include "hip/hip_runtime.h"
// This file is part of PlaneSweepLib (PSL)

// Copyright 2016 Christian Haene (ETH Zuerich)

// PSL is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.

// PSL is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.

// You should have received a copy of the GNU General Public License
// along with PSL.  If not, see <http://www.gnu.org/licenses/>.

#include "deviceBuffer.h"
#include "deviceBuffer.cuh"
#include <iostream>

namespace PSL_CUDA
{
    namespace DeviceBufferDeviceCode
    {
        template<typename T>
        __global__ void clearKernel(DeviceBuffer<T> buf, T value)
        {
            // get position of outupt
            unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
            unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

            if (x < buf.getWidth() && y < buf.getHeight())
            {
                buf(x,y) = value;
            }
        }
    }
}

using namespace PSL;
using namespace PSL_CUDA;
using namespace DeviceBufferDeviceCode;


template<typename T>
DeviceBuffer<T>::DeviceBuffer()
{
    addr = 0;
}

template<typename T>
void DeviceBuffer<T>::allocatePitched(int width, int height)
{
//   // to avoid memory leaks
//   deallocate();

   PSL_CUDA_CHECKED_CALL( hipMallocPitch(&addr, &pitch, width*sizeof(T), height); )
   this->width = width;
   this->height = height;
}

template<typename T>
void DeviceBuffer<T>::reallocatePitched(int width, int height)
{
    if (addr != 0)
    {
        if (width == this->width && height == this->height)
        {
            return;
        }
        deallocate();
    }
    allocatePitched(width, height);
}



template<typename T>
void DeviceBuffer<T>::deallocate()
{
//    if (addr != 0)
//    {
        PSL_CUDA_CHECKED_CALL( hipFree((void *)addr); )
//    }
    addr = 0;
}

template<typename T>
void DeviceBuffer<T>::clear(T value)
{

    dim3 gridDim(getNumTiles(width, TILE_WIDTH), getNumTiles(height, TILE_HEIGHT));
    dim3 blockDim(TILE_WIDTH, TILE_HEIGHT);

    clearKernel<<<gridDim, blockDim>>>(*this, value);
}

template<typename T>
void DeviceBuffer<T>::upload(T* dataPtr, size_t dataPitch)
{
    // pitch is in bytes
    PSL_CUDA_CHECKED_CALL( hipMemcpy2D(addr, pitch, dataPtr, dataPitch, width*sizeof(T), height, hipMemcpyHostToDevice); )
}


#ifdef _MSC_VER
#pragma warning( disable : 4661)
#endif

// instantiate needed buffers
template class DeviceBuffer<float>;
template class DeviceBuffer<int>;

