#include "hip/hip_runtime.h"
// This file is part of PlaneSweepLib (PSL)

// Copyright 2016 Christian Haene (ETH Zuerich)

// PSL is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.

// PSL is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.

// You should have received a copy of the GNU General Public License
// along with PSL.  If not, see <http://www.gnu.org/licenses/>.


#include <psl_cudaBase/cudaCommon.h>

#include <psl_cudaBase/deviceBuffer.h>
#include <psl_cudaBase/deviceBuffer.cuh>

#include <psl_cudaBase/deviceImage.h>
#include <psl_cudaBase/deviceImage.cuh>

#include <vector>


namespace PSL_CUDA
{
    namespace CudaFishEyePlaneSweepDeviceCode
    {
        __forceinline__ __device__ float computeWarpedGrayscaleTexturePixel(int x , int y, int width, int height,
                                                                            float krefinv11, float krefinv13, float krefinv22, float krefinv23, float xiref,
                                                                            float kother11, float kother13, float kother22, float kother23, float xiother,
                                                                            float rt11, float rt12, float rt13, float rt21, float rt22, float rt23, float rt31, float rt32, float rt33);
        __forceinline__ __device__ float computeZNCC(float normalizer, float ref, float refSqr, float other, float otherSqr, float prod);


        texture<uchar4, 2, hipReadModeNormalizedFloat> planeSweepColorTexture;
        texture<uchar1, 2, hipReadModeNormalizedFloat> planeSweepGrayscaleTexture;
        texture<unsigned char, 2> planeSweepGrayscaleTextureNonInterp;
        texture<float, 2> planeSweepLUTTexture;
        texture<float, 2> planeSweepCostTexture;

        hipChannelFormatDesc planeSweepColorChannelDesc;
        hipChannelFormatDesc planeSweepGrayscaleChannelDesc;
        hipChannelFormatDesc planeSweepCostChannelDesc;
        hipChannelFormatDesc planeSweepLUTChannelDesc;

        bool planeSweepTexturesInitialized = false;

        const int PLANE_SWEEP_TILE_WIDTH = 32;
        const int PLANE_SWEEP_TILE_HEIGHT = 8;

        const int PLANE_SWEEP_BOX_FILTER_NUM_THREADS = 128;
        const int PLANE_SWEEP_BOX_FILTER_ROWS_PER_THREAD = 25;

        void planeSweepInitTexturing()
        {
            if (!planeSweepTexturesInitialized)
            {
                // textures for image warping
                planeSweepColorChannelDesc = hipCreateChannelDesc(8,8,8,8,hipChannelFormatKindUnsigned);
                planeSweepGrayscaleChannelDesc = hipCreateChannelDesc(8,0,0,0,hipChannelFormatKindUnsigned);

                planeSweepColorTexture.addressMode[0] = hipAddressModeWrap;
                planeSweepColorTexture.addressMode[1] = hipAddressModeWrap;
                planeSweepColorTexture.filterMode = hipFilterModeLinear;
                planeSweepColorTexture.normalized = true;

                planeSweepGrayscaleTexture.addressMode[0] = hipAddressModeWrap;
                planeSweepGrayscaleTexture.addressMode[1] = hipAddressModeWrap;
                planeSweepGrayscaleTexture.filterMode = hipFilterModeLinear;
                planeSweepGrayscaleTexture.normalized = true;

                planeSweepGrayscaleTextureNonInterp.addressMode[0] = hipAddressModeWrap;
                planeSweepGrayscaleTextureNonInterp.addressMode[1] = hipAddressModeWrap;
                planeSweepGrayscaleTextureNonInterp.filterMode = hipFilterModePoint;
                planeSweepGrayscaleTextureNonInterp.normalized = false;

                planeSweepLUTTexture.addressMode[0] = hipAddressModeWrap;
                planeSweepLUTTexture.addressMode[1] = hipAddressModeWrap;
                planeSweepLUTTexture.filterMode = hipFilterModePoint;
                planeSweepLUTTexture.normalized = false;

                planeSweepLUTChannelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

                // textures for box filtering costs
                planeSweepCostChannelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

                planeSweepCostTexture.addressMode[0] = hipAddressModeClamp;
                planeSweepCostTexture.addressMode[1] = hipAddressModeClamp;
                planeSweepCostTexture.filterMode = hipFilterModePoint;
                planeSweepCostTexture.normalized = false;

                planeSweepTexturesInitialized = true;
            }
        }


        __global__ void boxFilterCostsKernel(DeviceBuffer<float> filteredBuf, int radius_x, int radius_y)
        {
            // implemented according to "Stereo Imaging with CUDA" and lots of changes since initial implementation
            extern __shared__ float colSum[];

            const int X =  blockIdx.x*PLANE_SWEEP_BOX_FILTER_NUM_THREADS + threadIdx.x - blockIdx.x*2*radius_x;
            const int Y =  blockIdx.y*PLANE_SWEEP_BOX_FILTER_ROWS_PER_THREAD;

            const int width = filteredBuf.getWidth();
            const int height = filteredBuf.getHeight();

            if (X < (width + 2*radius_x) && Y < height)
            {
                int x_cost = X - radius_x;
                colSum[threadIdx.x] = 0;

                int y_cost = Y - radius_y;
                for (int i = 0; i <= 2*radius_y; i++)
                {
                    colSum[threadIdx.x] += tex2D(planeSweepCostTexture, x_cost, y_cost);
                    y_cost++;
                }
                __syncthreads();


                if (threadIdx.x + 2*radius_x < PLANE_SWEEP_BOX_FILTER_NUM_THREADS && X < width && Y < height)
                {
                    float sum = 0;
                    for (int i = 0; i <= 2*radius_x; i++)
                    {
                        sum = sum + colSum[i+threadIdx.x];
                    }
                    filteredBuf(X, Y) = sum;
                }
                __syncthreads();

                y_cost = Y - radius_y;
                for (int row = 1; row < PLANE_SWEEP_BOX_FILTER_ROWS_PER_THREAD && (Y + row < height); row++)
                {
                    colSum[threadIdx.x] -= tex2D(planeSweepCostTexture, x_cost, y_cost);
                    colSum[threadIdx.x] += tex2D(planeSweepCostTexture, x_cost, y_cost + 2*radius_y + 1);

                    y_cost++;
                    __syncthreads();

                    if (threadIdx.x + 2*radius_x < PLANE_SWEEP_BOX_FILTER_NUM_THREADS && X < width)
                    {
                        float sum = 0;
                        for (int i = 0; i <= 2*radius_x; i++)
                        {
                            sum += colSum[i+threadIdx.x];
                        }
                        filteredBuf(X,Y+row) = sum;
                    }
                    __syncthreads();
                }
            }
        }

        void planeSweepBoxFilterCosts(DeviceBuffer<float>& costBuf, DeviceBuffer<float>& filteredCostBuf, int radius_x, int radius_y)
        {
            // Bind texture
            PSL_CUDA_CHECKED_CALL( hipBindTexture2D(0, planeSweepCostTexture, costBuf.getAddr(), planeSweepCostChannelDesc, costBuf.getWidth(), costBuf.getHeight(), costBuf.getPitch()); )

            const int sharedMemSize = PLANE_SWEEP_BOX_FILTER_NUM_THREADS*sizeof(float);

            dim3 gridDim(getNumTiles(filteredCostBuf.getWidth(), PLANE_SWEEP_BOX_FILTER_NUM_THREADS - 2*radius_x), getNumTiles(filteredCostBuf.getHeight(), PLANE_SWEEP_BOX_FILTER_ROWS_PER_THREAD));
            dim3 blockDim(PLANE_SWEEP_BOX_FILTER_NUM_THREADS, 1);

            // run box filtering kernel
            boxFilterCostsKernel<<<gridDim, blockDim, sharedMemSize>>>(filteredCostBuf, radius_x, radius_y);
            PSL_CUDA_CHECK_ERROR

            // unbind texture
            PSL_CUDA_CHECKED_CALL( hipUnbindTexture(planeSweepCostTexture); )
        }


        __global__ void planeSweepBoxFilterImageAndSqrImageKernel(DeviceBuffer<float> boxFilterBuf, DeviceBuffer<float> boxFilterSqrBuf, int radius_x, int radius_y)
        {
            // implemented according to "Stereo Imaging with CUDA" and lots of changes since the initial implementation

            extern __shared__ float colSum[];

            const int sqrOffset = PLANE_SWEEP_BOX_FILTER_NUM_THREADS;

            const int X = blockIdx.x*PLANE_SWEEP_BOX_FILTER_NUM_THREADS + threadIdx.x - blockIdx.x*2*radius_x;
            const int Y = blockIdx.y*PLANE_SWEEP_BOX_FILTER_ROWS_PER_THREAD;

            const int width = boxFilterBuf.getWidth();
            const int height = boxFilterBuf.getHeight();

            if (X < (width + 2*radius_x) && Y < height)
            {
                int x_img = X - radius_x;
                colSum[threadIdx.x] = 0;
                colSum[sqrOffset + threadIdx.x] = 0;

                int y_img = Y - radius_y;
                for (int i = 0; i <= 2*radius_y; i++)
                {
                    const float val = tex2D(planeSweepGrayscaleTextureNonInterp, x_img, y_img);
                    colSum[threadIdx.x] += val;
                    colSum[sqrOffset + threadIdx.x] += val*val;

                    y_img++;
                }
                __syncthreads();

                if (threadIdx.x + 2*radius_x < PLANE_SWEEP_BOX_FILTER_NUM_THREADS && X < width && Y < height)
                {
                    float sum = 0;
                    float sqrSum = 0;
                    for (int i = 0; i <= 2*radius_x; i++)
                    {
                        sum += colSum[i+threadIdx.x];
                        sqrSum += colSum[sqrOffset + i + threadIdx.x];
                    }
                    boxFilterBuf(X,Y) = sum;
                    boxFilterSqrBuf(X,Y) = sqrSum;
                }

                __syncthreads();

                y_img = Y - radius_y;
                for (int row = 1; row < PLANE_SWEEP_BOX_FILTER_ROWS_PER_THREAD && (row + Y) < height; row++)
                {
                    const float subVal = tex2D(planeSweepGrayscaleTextureNonInterp, x_img, y_img);
                    colSum[threadIdx.x] -= subVal;
                    colSum[sqrOffset + threadIdx.x] -= subVal*subVal;
                    const float addVal = tex2D(planeSweepGrayscaleTextureNonInterp, x_img, y_img + 2*radius_y + 1);
                    colSum[threadIdx.x] += addVal;
                    colSum[sqrOffset + threadIdx.x] += addVal*addVal;
                    y_img++;
                    __syncthreads();

                    if (threadIdx.x + 2*radius_x < PLANE_SWEEP_BOX_FILTER_NUM_THREADS && X < width)
                    {
                        float sum = 0;
                        float sqrSum = 0;
                        for (int i = 0; i <= 2*radius_x; i++)
                        {
                            sum += colSum[i+threadIdx.x];
                            sqrSum += colSum[sqrOffset + i + threadIdx.x];
                        }
                        boxFilterBuf(X,Y+row) = sum;
                        boxFilterSqrBuf(X,Y+row) = sqrSum;
                    }
                    __syncthreads();
                }
            }
        }

        void planeSweepBoxFilterImageAndSqrImage(const DeviceImage& refImg,
                                                 DeviceBuffer<float>& boxFilterBuf, DeviceBuffer<float>& boxFilterSqrBuf,
                                                 DeviceBuffer<float>& tempBuf, DeviceBuffer<float>& tempSqrBuf, int radius_x, int radius_y)
        {

            const int width = refImg.getWidth();
            const int height = refImg.getHeight();

            // bind texture
            PSL_CUDA_CHECKED_CALL( hipBindTexture2D(0, planeSweepGrayscaleTextureNonInterp, refImg.getAddr(), planeSweepGrayscaleChannelDesc, refImg.getWidth(), refImg.getHeight(), refImg.getPitch()); )

            const int sharedMemSize = 2*(PLANE_SWEEP_BOX_FILTER_NUM_THREADS)*sizeof(float);

            dim3 gridDim(getNumTiles(width, PLANE_SWEEP_BOX_FILTER_NUM_THREADS - 2*radius_x), getNumTiles(height, PLANE_SWEEP_BOX_FILTER_ROWS_PER_THREAD));
            dim3 blockDim(PLANE_SWEEP_BOX_FILTER_NUM_THREADS, 1);
            planeSweepBoxFilterImageAndSqrImageKernel<<<gridDim, blockDim, sharedMemSize>>>(boxFilterBuf, boxFilterSqrBuf, radius_x, radius_y);
            PSL_CUDA_CHECK_ERROR

            PSL_CUDA_CHECKED_CALL(hipUnbindTexture(planeSweepGrayscaleTextureNonInterp))
        }



        __global__ void planeSweepZNCCKernel(float krefinv11, float krefinv13, float krefinv22, float krefinv23, float xiref,
                                             float kother11, float kother13, float kother22, float kother23, float xiother,
                                             float rt11, float rt12, float rt13, float rt21, float rt22, float rt23, float rt31, float rt32, float rt33,
                                             DeviceBuffer<float> refFiltered, DeviceBuffer<float> refSqrFiltered,
                                             DeviceBuffer<float> costBuffer, int radius_x, int radius_y, float normalizerZNCC)
        {
            // implemented according to "Stereo Imaging with CUDA" and lots of changes since initial implementation

            const int sqrOffset = PLANE_SWEEP_BOX_FILTER_NUM_THREADS;
            const int prodOffset = 2*sqrOffset;

            extern __shared__ float colSum[];

            const int X = (blockIdx.x*PLANE_SWEEP_BOX_FILTER_NUM_THREADS + threadIdx.x)-blockIdx.x*2*radius_x;
            const int Y = blockIdx.y*PLANE_SWEEP_BOX_FILTER_ROWS_PER_THREAD;

            const int width = costBuffer.getWidth();
            const int height = costBuffer.getHeight();

            if (X < (width + 2*radius_x) && Y < height)
            {
                int x_img = X - radius_x;
                colSum[threadIdx.x] = 0;
                colSum[sqrOffset + threadIdx.x] = 0;
                colSum[prodOffset + threadIdx.x] = 0;

                int y_img = Y - radius_y;
                for (int i = 0; i <= 2*radius_y; i++)
                {
                    const float val = tex2D(planeSweepGrayscaleTextureNonInterp, x_img, y_img);
                    const float otherVal = computeWarpedGrayscaleTexturePixel(x_img, y_img, width, height,
                                                                              krefinv11, krefinv13, krefinv22, krefinv23, xiref,
                                                                              kother11, kother13, kother22, kother23, xiother,
                                                                              rt11, rt12, rt13, rt21, rt22, rt23, rt31, rt32, rt33);
                    colSum[threadIdx.x] += otherVal;
                    colSum[sqrOffset + threadIdx.x] += otherVal*otherVal;
                    colSum[prodOffset + threadIdx.x] += val*otherVal;

                    y_img++;
                }
                __syncthreads();

                if (threadIdx.x + 2*radius_x < PLANE_SWEEP_BOX_FILTER_NUM_THREADS && X < width && Y < height)
                {
                    float other = 0;
                    float otherSqr = 0;
                    float prod = 0;
                    for (int i = 0; i <= 2*radius_x; i++)
                    {
                        other += colSum[i+threadIdx.x];
                        otherSqr += colSum[sqrOffset + i + threadIdx.x];
                        prod += colSum[prodOffset + i + threadIdx.x];
                    }
                    const float ref = refFiltered(X,Y);
                    const float refSqr = refSqrFiltered(X,Y);

                    costBuffer(X,Y) = computeZNCC(normalizerZNCC, ref, refSqr, other, otherSqr, prod);
                }

                __syncthreads();

                y_img = Y - radius_y;
                for (int row = 1; row < PLANE_SWEEP_BOX_FILTER_ROWS_PER_THREAD && (row + Y < height); row++)
                {
                    const float subVal = tex2D(planeSweepGrayscaleTextureNonInterp, x_img, y_img);
                    const float subOtherVal = computeWarpedGrayscaleTexturePixel(x_img, y_img, width, height,
                                                                                 krefinv11, krefinv13, krefinv22, krefinv23, xiref,
                                                                                 kother11, kother13, kother22, kother23, xiother,
                                                                                 rt11, rt12, rt13, rt21, rt22, rt23, rt31, rt32, rt33);
                    colSum[threadIdx.x] -= subOtherVal;
                    colSum[sqrOffset + threadIdx.x] -= subOtherVal*subOtherVal;
                    colSum[prodOffset + threadIdx.x] -= subOtherVal*subVal;
                    const float addVal = tex2D(planeSweepGrayscaleTextureNonInterp, x_img, y_img + 2*radius_y + 1);
                    const float addOtherVal = computeWarpedGrayscaleTexturePixel( x_img, y_img + 2*radius_y + 1, width, height,
                                                                                  krefinv11, krefinv13, krefinv22, krefinv23, xiref,
                                                                                  kother11, kother13, kother22, kother23, xiother,
                                                                                  rt11, rt12, rt13, rt21, rt22, rt23, rt31, rt32, rt33);
                    colSum[threadIdx.x] += addOtherVal;
                    colSum[sqrOffset + threadIdx.x] += addOtherVal*addOtherVal;
                    colSum[prodOffset + threadIdx.x] += addOtherVal*addVal;
                    y_img++;

                    __syncthreads();

                    if (threadIdx.x + 2*radius_x < PLANE_SWEEP_BOX_FILTER_NUM_THREADS && X < width)
                    {
                        float other = 0;
                        float otherSqr = 0;
                        float prod = 0;
                        for (int i = 0; i <= 2*radius_x; i++)
                        {
                            other += colSum[i+threadIdx.x];
                            otherSqr += colSum[sqrOffset + i + threadIdx.x];
                            prod += colSum[prodOffset + i + threadIdx.x];
                        }
                        const float ref = refFiltered(X,Y+row);
                        const float refSqr = refSqrFiltered(X,Y+row);

                        costBuffer(X, Y+row) = computeZNCC(normalizerZNCC, ref, refSqr, other, otherSqr, prod);
                    }
                    __syncthreads();
                }
            }
        }

        void planeSweepWarpZNCC(const DeviceImage &otherImg, float* RT,
                                const float* KRefInv, const float XiRef,
                                const float* Kother, const float XiOther,
                                const DeviceImage &refImg,
                                DeviceBuffer<float>& refImgBoxFilterBuf, DeviceBuffer<float>& refImgSqrBoxFilterBuf,
                                DeviceBuffer<float>& costBuf, int radius_x, int radius_y)
        {
            // bind textures
            PSL_CUDA_CHECKED_CALL( hipBindTexture2D(0, planeSweepGrayscaleTexture, otherImg.getAddr(), planeSweepGrayscaleChannelDesc, otherImg.getWidth(), otherImg.getHeight(), otherImg.getPitch()); )
            PSL_CUDA_CHECKED_CALL( hipBindTexture2D(0, planeSweepGrayscaleTextureNonInterp, refImg.getAddr(), planeSweepGrayscaleChannelDesc, refImg.getWidth(), refImg.getHeight(), refImg.getPitch()); )

            // compute kernel config
            const int sharedMemSize = 3*(PLANE_SWEEP_BOX_FILTER_NUM_THREADS)*sizeof(float);
            dim3 gridDim(getNumTiles(refImg.getWidth(), PLANE_SWEEP_BOX_FILTER_NUM_THREADS-2*radius_x), getNumTiles(refImg.getHeight(), PLANE_SWEEP_BOX_FILTER_ROWS_PER_THREAD));
            dim3 blockDim(PLANE_SWEEP_BOX_FILTER_NUM_THREADS, 1);

            const float normalizerZNCC = 1.0f/((float)(2*radius_x+1)*(2*radius_y+1));

            // run filter kernel
            planeSweepZNCCKernel<<<gridDim, blockDim, sharedMemSize>>>(KRefInv[0], KRefInv[2], KRefInv[4], KRefInv[5], XiRef,
                                                                       Kother[0], Kother[2], Kother[4], Kother[5], XiOther,
                                                                       RT[0], RT[1], RT[2], RT[3], RT[4], RT[5], RT[6], RT[7], RT[8],
                                                                       refImgBoxFilterBuf, refImgSqrBoxFilterBuf,
                                                                       costBuf, radius_x, radius_y, normalizerZNCC);
            PSL_CUDA_CHECK_ERROR

            // unbind textures
            PSL_CUDA_CHECKED_CALL( hipUnbindTexture(planeSweepGrayscaleTexture) )
            PSL_CUDA_CHECKED_CALL( hipUnbindTexture(planeSweepGrayscaleTextureNonInterp) )
        }



        __global__ void planeSweepZNCCAccumKernel(float krefinv11, float krefinv13, float krefinv22, float krefinv23, float xiref,
                                                  float kother11, float kother13, float kother22, float kother23, float xiother,
                                                  float rt11, float rt12, float rt13, float rt21, float rt22, float rt23, float rt31, float rt32, float rt33,
                                                  DeviceBuffer<float> refFiltered, DeviceBuffer<float> refSqrFiltered,
                                                  DeviceBuffer<float> costAccumBuffer, float accumScale, int radius_x, int radius_y, float normalizerZNCC)
        {
            // implemented according to "Stereo Imaging with CUDA" and lots of changes since initial implementation

            const int sqrOffset = PLANE_SWEEP_BOX_FILTER_NUM_THREADS;
            const int prodOffset = 2*sqrOffset;

            extern __shared__ float colSum[];

            const int X = (blockIdx.x*PLANE_SWEEP_BOX_FILTER_NUM_THREADS + threadIdx.x)-blockIdx.x*2*radius_x;
            const int Y = blockIdx.y*PLANE_SWEEP_BOX_FILTER_ROWS_PER_THREAD;

            const int width = costAccumBuffer.getWidth();
            const int height = costAccumBuffer.getHeight();

            if (X < (width + 2*radius_x) && Y < height)
            {
                int x_img = X - radius_x;
                colSum[threadIdx.x] = 0;
                colSum[sqrOffset + threadIdx.x] = 0;
                colSum[prodOffset + threadIdx.x] = 0;

                int y_img = Y - radius_y;
                for (int i = 0; i <= 2*radius_y; i++)
                {
                    const float val = tex2D(planeSweepGrayscaleTextureNonInterp, x_img, y_img);
                    const float otherVal = computeWarpedGrayscaleTexturePixel(x_img, y_img, width, height,
                                                                              krefinv11, krefinv13, krefinv22, krefinv23, xiref,
                                                                              kother11, kother13, kother22, kother23, xiother,
                                                                              rt11, rt12, rt13, rt21, rt22, rt23, rt31, rt32, rt33);
                    colSum[threadIdx.x] += otherVal;
                    colSum[sqrOffset + threadIdx.x] += otherVal*otherVal;
                    colSum[prodOffset + threadIdx.x] += val*otherVal;

                    y_img++;
                }
                __syncthreads();

                if (threadIdx.x + 2*radius_x < PLANE_SWEEP_BOX_FILTER_NUM_THREADS && X < width && Y < height)
                {
                    float other = 0;
                    float otherSqr = 0;
                    float prod = 0;
                    for (int i = 0; i <= 2*radius_x; i++)
                    {
                        other += colSum[i+threadIdx.x];
                        otherSqr += colSum[sqrOffset + i + threadIdx.x];
                        prod += colSum[prodOffset + i + threadIdx.x];
                    }
                    const float ref = refFiltered(X,Y);
                    const float refSqr = refSqrFiltered(X,Y);

                    costAccumBuffer(X,Y) += accumScale*computeZNCC(normalizerZNCC, ref, refSqr, other, otherSqr, prod);
                }
                __syncthreads();

                y_img = Y - radius_y;
                for (int row = 1; row < PLANE_SWEEP_BOX_FILTER_ROWS_PER_THREAD && (row + Y < height); row++)
                {
                    const float subVal = tex2D(planeSweepGrayscaleTextureNonInterp, x_img, y_img);
                    const float subOtherVal = computeWarpedGrayscaleTexturePixel(x_img, y_img, width, height,
                                                                                 krefinv11, krefinv13, krefinv22, krefinv23, xiref,
                                                                                 kother11, kother13, kother22, kother23, xiother,
                                                                                 rt11, rt12, rt13, rt21, rt22, rt23, rt31, rt32, rt33);
                    colSum[threadIdx.x] -= subOtherVal;
                    colSum[sqrOffset + threadIdx.x] -= subOtherVal*subOtherVal;
                    colSum[prodOffset + threadIdx.x] -= subOtherVal*subVal;
                    const float addVal = tex2D(planeSweepGrayscaleTextureNonInterp, x_img, y_img + 2*radius_y + 1);
                    const float addOtherVal = computeWarpedGrayscaleTexturePixel( x_img, y_img + 2*radius_y + 1, width, height,
                                                                                  krefinv11, krefinv13, krefinv22, krefinv23, xiref,
                                                                                  kother11, kother13, kother22, kother23, xiother,
                                                                                  rt11, rt12, rt13, rt21, rt22, rt23, rt31, rt32, rt33);
                    colSum[threadIdx.x] += addOtherVal;
                    colSum[sqrOffset + threadIdx.x] += addOtherVal*addOtherVal;
                    colSum[prodOffset + threadIdx.x] += addOtherVal*addVal;
                    y_img++;

                    __syncthreads();

                    if (threadIdx.x + 2*radius_x < PLANE_SWEEP_BOX_FILTER_NUM_THREADS && X < width)
                    {
                        float other = 0;
                        float otherSqr = 0;
                        float prod = 0;
                        for (int i = 0; i <= 2*radius_x; i++)
                        {
                            other += colSum[i+threadIdx.x];
                            otherSqr += colSum[sqrOffset + i + threadIdx.x];
                            prod += colSum[prodOffset + i + threadIdx.x];
                        }
                        const float ref = refFiltered(X,Y+row);
                        const float refSqr = refSqrFiltered(X,Y+row);

                        costAccumBuffer(X, Y+row) += accumScale*computeZNCC(normalizerZNCC, ref, refSqr, other, otherSqr, prod);
                    }
                    __syncthreads();
                }
            }
        }

        void planeSweepWarpZNCCAccum(const DeviceImage& otherImg, float* RT,
                                     const float* KRefInv, const float XiRef,
                                     const float* Kother, const float XiOther,
                                     const DeviceImage& refImg,
                                     DeviceBuffer<float>& refImgBoxFilterBuf, DeviceBuffer<float>& refImgSqrBoxFilterBuf,
                                     float accumScale, DeviceBuffer<float>& costAccumBuf, int radius_x, int radius_y)
        {
            // bind textures
            PSL_CUDA_CHECKED_CALL( hipBindTexture2D(0, planeSweepGrayscaleTexture, otherImg.getAddr(), planeSweepGrayscaleChannelDesc, otherImg.getWidth(), otherImg.getHeight(), otherImg.getPitch()); )
            PSL_CUDA_CHECKED_CALL( hipBindTexture2D(0, planeSweepGrayscaleTextureNonInterp, refImg.getAddr(), planeSweepGrayscaleChannelDesc, refImg.getWidth(), refImg.getHeight(), refImg.getPitch()); )

            // compute kernel config
            const int sharedMemSize = 3*(PLANE_SWEEP_BOX_FILTER_NUM_THREADS)*sizeof(float);
            dim3 gridDim(getNumTiles(refImg.getWidth(), PLANE_SWEEP_BOX_FILTER_NUM_THREADS-2*radius_x), getNumTiles(refImg.getHeight(), PLANE_SWEEP_BOX_FILTER_ROWS_PER_THREAD));
            dim3 blockDim(PLANE_SWEEP_BOX_FILTER_NUM_THREADS, 1);

            const float normalizerZNCC = 1.0f/((float)(2*radius_x+1)*(2*radius_y+1));

            // run filter kernel
            planeSweepZNCCAccumKernel<<<gridDim, blockDim, sharedMemSize>>>(KRefInv[0], KRefInv[2], KRefInv[4], KRefInv[5], XiRef,
                                                                            Kother[0], Kother[2], Kother[4], Kother[5], XiOther,
                                                                            RT[0], RT[1], RT[2], RT[3], RT[4], RT[5], RT[6], RT[7], RT[8],
                                                                            refImgBoxFilterBuf, refImgSqrBoxFilterBuf,
                                                                            costAccumBuf, accumScale, radius_x, radius_y, normalizerZNCC);
            PSL_CUDA_CHECK_ERROR

            // unbind textures
            PSL_CUDA_CHECKED_CALL( hipUnbindTexture(planeSweepGrayscaleTexture) )
            PSL_CUDA_CHECKED_CALL( hipUnbindTexture(planeSweepGrayscaleTextureNonInterp) )
        }


        __global__ void planeSweepWarpADGrayscaleKernel(const int srcImgWidth, const int srcImgHeight,
                                                        float krefinv11, float krefinv13, float krefinv22, float krefinv23, float xiref,
                                                        float kother11, float kother13, float kother22, float kother23, float xiother,
                                                        float rt11, float rt12, float rt13, float rt21, float rt22, float rt23, float rt31, float rt32, float rt33,
                                                        DeviceImage refImg, DeviceBuffer<float> costBuf)
        {
            // get position of outupt
            const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
            const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

            const int width = refImg.getWidth();
            const int height = refImg.getHeight();

            if (x < width && y < height)
            {
                // apply Krefinv
                float mx = krefinv11*x + krefinv13;
                float my = krefinv22*y + krefinv23;

                // apply h^-1 of camera model
                float mxPmySqr = mx*mx + my*my;
                float D = 1 + (1 - xiref*xiref)*mxPmySqr;

                if (D < 0)
                    D = -D;

                float fact = (xiref + sqrtf(D))/(mxPmySqr + 1);

                float xx = fact*mx;
                float yy = fact*my;
                float zz = fact - xiref;

                // apply rt
                float xxw = rt11*xx + rt12*yy + rt13*zz;
                float yyw = rt21*xx + rt22*yy + rt23*zz;
                float zzw = rt31*xx + rt32*yy + rt33*zz;

                // apply h
                float length = sqrtf(xxw*xxw + yyw*yyw + zzw*zzw);
                float xxxw = xxw/length;
                float yyyw = yyw/length;
                float zzzw = zzw/length + xiother;

                float xmw = xxxw/zzzw;
                float ymw = yyyw/zzzw;

                // apply Kother

                float xw = kother11*xmw + kother13;
                float yw = kother22*ymw + kother23;

                const float u = (xw+0.5f) / (float) srcImgWidth;
                const float v = (yw+0.5f) / (float) srcImgHeight;

                const float1 pix = tex2D(planeSweepGrayscaleTexture, u, v);

                const float i = __saturatef(fabs(pix.x))*255;

                // ad
                float id = fabs((float)(refImg(x,y) - i));
//                float id = fabs((float)(i));

                // accumulate
                costBuf(x,y) = id;
            }
        }

        void planeSweepWarpAD(const DeviceImage &srcImg, const float* RT,
                              const float* KRefInv, const float XiRef,
                              const float* Kother, const float XiOther,
                              const DeviceImage &refImg, DeviceBuffer<float>& costBuf)
        {
            dim3 gridDim(getNumTiles(refImg.getWidth(), PLANE_SWEEP_TILE_WIDTH), getNumTiles(refImg.getHeight(), PLANE_SWEEP_TILE_HEIGHT));
            dim3 blockDim(PLANE_SWEEP_TILE_WIDTH, PLANE_SWEEP_TILE_HEIGHT);

            // bind the grayscale texture
            PSL_CUDA_CHECKED_CALL( hipBindTexture2D(0, planeSweepGrayscaleTexture, srcImg.getAddr(), planeSweepGrayscaleChannelDesc, srcImg.getWidth(), srcImg.getHeight(), srcImg.getPitch()); )

            planeSweepWarpADGrayscaleKernel<<<gridDim, blockDim>>>(srcImg.getWidth(), srcImg.getHeight(),
                                                                   KRefInv[0], KRefInv[2], KRefInv[4], KRefInv[5], XiRef,
                                                                   Kother[0], Kother[2], Kother[4], Kother[5], XiOther,
                                                                   RT[0], RT[1], RT[2], RT[3], RT[4], RT[5], RT[6], RT[7], RT[8],
                                                                   refImg, costBuf);
            PSL_CUDA_CHECK_ERROR
            PSL_CUDA_CHECKED_CALL( hipUnbindTexture(planeSweepGrayscaleTexture); )
        }



        __global__ void planeSweepWarpADAccumGrayscaleKernel(const int srcImgWidth, const int srcImgHeight,
                                                             float krefinv11, float krefinv13, float krefinv22, float krefinv23, float xiref,
                                                             float kother11, float kother13, float kother22, float kother23, float xiother,
                                                             float rt11, float rt12, float rt13, float rt21, float rt22, float rt23, float rt31, float rt32, float rt33,
                                                             DeviceImage refImg, float accumScale, DeviceBuffer<float> costAccumBuf)
        {
            // get position of outupt
            const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
            const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

            const int width = refImg.getWidth();
            const int height = refImg.getHeight();

            if (x < width && y < height)
            {
                // apply Krefinv
                float mx = krefinv11*x + krefinv13;
                float my = krefinv22*y + krefinv23;

                // apply h^-1 of camera model
                float mxPmySqr = mx*mx + my*my;
                float D = 1 + (1 - xiref*xiref)*mxPmySqr;

                if (D < 0)
                    D = -D;

                float fact = (xiref + sqrtf(D))/(mxPmySqr + 1);

                float xx = fact*mx;
                float yy = fact*my;
                float zz = fact - xiref;

                // apply rt
                float xxw = rt11*xx + rt12*yy + rt13*zz;
                float yyw = rt21*xx + rt22*yy + rt23*zz;
                float zzw = rt31*xx + rt32*yy + rt33*zz;

                // apply h
                float length = sqrtf(xxw*xxw + yyw*yyw + zzw*zzw);
                float xxxw = xxw/length;
                float yyyw = yyw/length;
                float zzzw = zzw/length + xiother;

                float xmw = xxxw/zzzw;
                float ymw = yyyw/zzzw;

                // apply Kother

                float xw = kother11*xmw + kother13;
                float yw = kother22*ymw + kother23;

                const float u = (xw+0.5f) / (float) srcImgWidth;
                const float v = (yw+0.5f) / (float) srcImgHeight;

                const float1 pix = tex2D(planeSweepGrayscaleTexture, u, v);

                const float i = __saturatef(fabs(pix.x))*255;

                // ad
                float id = fabs((float)(refImg(x,y) - i));
//                float id = fabs((float)(i));

                // accumulate
                costAccumBuf(x,y) += accumScale*id;
            }
        }

        void planeSweepWarpADAccum(DeviceImage& srcImg, const float* RT,
                                   const float* KRefInv, const float XiRef,
                                   const float* Kother, const float XiOther,
                                   DeviceImage& refImg,
                                   float accumScale, DeviceBuffer<float>& costAccumBuf)
        {
            dim3 gridDim(getNumTiles(refImg.getWidth(), PLANE_SWEEP_TILE_WIDTH), getNumTiles(refImg.getHeight(), PLANE_SWEEP_TILE_HEIGHT));
            dim3 blockDim(PLANE_SWEEP_TILE_WIDTH, PLANE_SWEEP_TILE_HEIGHT);

            // bind the grayscale texture
            PSL_CUDA_CHECKED_CALL( hipBindTexture2D(0, planeSweepGrayscaleTexture, srcImg.getAddr(), planeSweepGrayscaleChannelDesc, srcImg.getWidth(), srcImg.getHeight(), srcImg.getPitch()); )

            planeSweepWarpADAccumGrayscaleKernel<<<gridDim, blockDim>>>(srcImg.getWidth(), srcImg.getHeight(),
                                                                        KRefInv[0], KRefInv[2], KRefInv[4], KRefInv[5], XiRef,
                                                                        Kother[0], Kother[2], Kother[4], Kother[5], XiOther,
                                                                        RT[0], RT[1], RT[2], RT[3], RT[4], RT[5], RT[6], RT[7], RT[8],
                                                                     refImg, accumScale, costAccumBuf);
            PSL_CUDA_CHECK_ERROR
            PSL_CUDA_CHECKED_CALL( hipUnbindTexture(planeSweepGrayscaleTexture); )
        }


        __global__ void planeSweepUpdateBestKKernel(DeviceBuffer<float> newCosts, DeviceBuffer<float> bestCosts, DeviceBuffer<float> bestMin)
        {
            unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
            unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

            if (x < newCosts.getWidth() && y < newCosts.getHeight())
            {
                const float newCost = newCosts(x,y);

                if (newCost > bestMin(x,y) && newCost < bestCosts(x,y))
                {
                    bestCosts(x,y) = newCost;
                }
            }
        }

        void updateBestK(DeviceBuffer<float>& newCostsBuf, DeviceBuffer<float>& bestCostsBuf, DeviceBuffer<float>& bestMinBuf)
        {
            dim3 gridDim(getNumTiles(bestCostsBuf.getWidth(), PLANE_SWEEP_TILE_WIDTH), getNumTiles(bestCostsBuf.getHeight(), PLANE_SWEEP_TILE_HEIGHT));
            dim3 blockDim(PLANE_SWEEP_TILE_WIDTH, PLANE_SWEEP_TILE_HEIGHT);

            planeSweepUpdateBestKKernel<<<gridDim, blockDim>>>(newCostsBuf, bestCostsBuf, bestMinBuf);
            PSL_CUDA_CHECK_ERROR
        }

        __global__ void planeSweepAccumCostKernelBestK(DeviceBuffer<float> accumBuf, DeviceBuffer<float> costBuf, DeviceBuffer<float> minCostBuf, float maxVal, float accumScale)
        {
            // get position of outupt
            const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
            const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

            if (x < accumBuf.getWidth() && y < accumBuf.getHeight())
            {
                const float newVal = costBuf(x,y);
                if (newVal < maxVal)
                {
                    accumBuf(x,y) += accumScale*newVal;
                }
                else
                {
                    const float usedVal = minCostBuf(x,y);
                    accumBuf(x,y) += accumScale*usedVal;
                    costBuf(x,y) = usedVal;
                }
            }

        }

        void planeSweepAccumCostBestK(DeviceBuffer<float>& costAccumBuf, DeviceBuffer<float>& costBuf, DeviceBuffer<float>& minCostBuf, float maxVal, float accumScale)
        {
            dim3 gridDim(getNumTiles(costAccumBuf.getWidth(), PLANE_SWEEP_TILE_WIDTH), getNumTiles(costAccumBuf.getHeight(), PLANE_SWEEP_TILE_HEIGHT));
            dim3 blockDim(PLANE_SWEEP_TILE_WIDTH, PLANE_SWEEP_TILE_HEIGHT);

            planeSweepAccumCostKernelBestK<<<gridDim, blockDim>>>(costAccumBuf, costBuf, minCostBuf, maxVal, accumScale);
            PSL_CUDA_CHECK_ERROR
        }

        __global__ void planeSweepUpdateBestPlaneKernel(DeviceBuffer<float> newCosts, int currPlaneIndex, DeviceBuffer<float> bestPlaneCosts, DeviceBuffer<int> bestPlanes)
        {
            // get position of outupt
            unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
            unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

            if (x < newCosts.getWidth() && y < newCosts.getHeight())
            {
                if (newCosts(x,y) <= bestPlaneCosts(x,y))
                {
                    bestPlaneCosts(x,y) = newCosts(x,y);
                    bestPlanes(x,y) = currPlaneIndex;
                }
            }

        }

        void planeSweepUpdateBestPlane(const DeviceBuffer<float>& newCosts, int width, int height, int currPlaneIndex,
                                       DeviceBuffer<float>& bestPlaneCosts, DeviceBuffer<int>& bestPlanes)
        {
            dim3 gridDim(getNumTiles(width, PLANE_SWEEP_TILE_WIDTH), getNumTiles(height, PLANE_SWEEP_TILE_HEIGHT));
            dim3 blockDim(PLANE_SWEEP_TILE_WIDTH, PLANE_SWEEP_TILE_HEIGHT);

            planeSweepUpdateBestPlaneKernel<<<gridDim, blockDim>>>(newCosts, currPlaneIndex, bestPlaneCosts, bestPlanes);
            PSL_CUDA_CHECK_ERROR
        }

        __global__ void planeSweepUpdateBestPlaneSubPixelKernel(DeviceBuffer<float> currCosts, DeviceBuffer<float> prev1, DeviceBuffer<float> prev2, int prevPlaneIdx,
                                                                DeviceBuffer<float> bestPlaneCosts, DeviceBuffer<int> bestPlanes, DeviceBuffer<float> subPixelPlaneOffsets)
        {
            // get position of outupt
            const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
            const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

            if (x < currCosts.getWidth() && y < currCosts.getHeight())
            {
                const float prev1_t = prev1(x,y);
                const float prev2_t = prev2(x,y);
                const float currCosts_t = currCosts(x,y);


                // we only comput the parabola if we found a new minimal cost
                if (prev1_t <= bestPlaneCosts(x,y) && prev2_t > prev1(x,y) && currCosts_t > prev1(x,y))
                {
                    // parabola
                    const float denom = currCosts(x,y) + prev2(x,y) - 2*prev1(x,y);
                    float offset = 0.0f;
                    if (denom > 1e-5f)
                    {
                        offset = (prev2(x,y) - prev1(x,y))/denom - 0.5f;
                    }

                    bestPlanes(x,y) = prevPlaneIdx;
                    subPixelPlaneOffsets(x,y) = offset;
                    bestPlaneCosts(x,y) = prev1(x,y); // we use the actual cost of the minimal plane for robustnes
                }
            }

        }

        void planeSweepUpdateBestPlaneSubPixel(const DeviceBuffer<float>& currCosts, const DeviceBuffer<float>& prev1, const DeviceBuffer<float>& prev2,
                                               int width, int height, int prevPlaneIdx, DeviceBuffer<float>& bestPlaneCosts, DeviceBuffer<int>& bestPlanes, DeviceBuffer<float>& subPixelPlaneOffsets)
        {
            dim3 gridDim(getNumTiles(width, PLANE_SWEEP_TILE_WIDTH), getNumTiles(height, PLANE_SWEEP_TILE_HEIGHT));
            dim3 blockDim(PLANE_SWEEP_TILE_WIDTH, PLANE_SWEEP_TILE_HEIGHT);

            planeSweepUpdateBestPlaneSubPixelKernel<<<gridDim, blockDim>>>(currCosts, prev1, prev2, prevPlaneIdx, bestPlaneCosts, bestPlanes, subPixelPlaneOffsets);
            PSL_CUDA_CHECK_ERROR
        }

        __global__ void planeSweepUpdateBestAndSecondBestPlaneKernel(DeviceBuffer<float> newCosts, int currPlaneIndex, DeviceBuffer<float> bestPlaneCosts,
                                                                     DeviceBuffer<float> secondBestPlaneCosts, DeviceBuffer<int> bestPlanes)
        {
            // get position of outupt
            unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
            unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

            if (x < newCosts.getWidth() && y < newCosts.getHeight())
            {

                if (newCosts(x,y) <= bestPlaneCosts(x,y))
                {
                    secondBestPlaneCosts(x,y) = bestPlaneCosts(x,y);
                    bestPlaneCosts(x,y) = newCosts(x,y);
                    bestPlanes(x,y) = currPlaneIndex;
                }
                else if(newCosts(x,y) <= secondBestPlaneCosts(x,y))
                {
                    secondBestPlaneCosts(x,y) = newCosts(x,y);
                }
            }

        }

        void planeSweepUpdateBestAndSecondBestPlane(const DeviceBuffer<float>& newCosts, int width, int height, int currPlaneIndex,
                                                    DeviceBuffer<float>& bestPlaneCosts, DeviceBuffer<float>& secondBestPlaneCosts, DeviceBuffer<int>& bestPlanes)
        {
            dim3 gridDim(getNumTiles(width, PLANE_SWEEP_TILE_WIDTH), getNumTiles(height, PLANE_SWEEP_TILE_HEIGHT));
            dim3 blockDim(PLANE_SWEEP_TILE_WIDTH, PLANE_SWEEP_TILE_HEIGHT);

            planeSweepUpdateBestAndSecondBestPlaneKernel<<<gridDim, blockDim>>>(newCosts, currPlaneIndex, bestPlaneCosts,
                                                                                secondBestPlaneCosts, bestPlanes);
            PSL_CUDA_CHECK_ERROR
        }

        __global__ void planeSweepUpdateBestAndSecondBestPlaneSubPixelKernel(DeviceBuffer<float> currCosts, DeviceBuffer<float> prev1, DeviceBuffer<float> prev2, int prevPlaneIdx,
                                                                             DeviceBuffer<float> bestPlaneCosts, DeviceBuffer<float> secondBestPlaneCosts, DeviceBuffer<int> bestPlanes, DeviceBuffer<float> subPixelPlaneOffsets)
        {
            // get position of outupt
            unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
            unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

            if (x < currCosts.getWidth() && y < currCosts.getHeight())
            {
                const float prev1_t = prev1(x,y);
                const float prev2_t = prev2(x,y);
                const float currCosts_t = currCosts(x,y);

                // we only comput the parabola if we found a new minimal cost
                if (prev1_t <= bestPlaneCosts(x,y) && prev2_t > prev1_t && currCosts_t > prev1_t)
                {
                    // parabola
                    const float denom = currCosts(x,y) + prev2(x,y) - 2*prev1(x,y);
                    float offset = 0.0f;
                    if (denom > 1e-5f)
                    {
                        offset = (prev2(x,y) - prev1(x,y))/denom - 0.5f;
                    }

                    bestPlanes(x,y) = prevPlaneIdx;
                    subPixelPlaneOffsets(x,y) = offset;
                    bestPlaneCosts(x,y) = prev1(x,y); // we use the actual cost of the minimal plane for robustnes
                }
                else if (prev1(x,y) <= secondBestPlaneCosts(x,y))
                {
                    secondBestPlaneCosts(x,y) = prev1(x,y);
                }
            }
        }


        void planeSweepUpdateBestAndSecondBestPlaneSubPixel(const DeviceBuffer<float>& currCosts, const DeviceBuffer<float>& prev1, const DeviceBuffer<float>& prev2,
                                                            int width, int height, int prevPlaneIdx, DeviceBuffer<float>& bestPlaneCosts, DeviceBuffer<float>& secondBestPlaneCosts, DeviceBuffer<int>& bestPlanes, DeviceBuffer<float>& subPixelPlaneOffsets)

        {
            dim3 gridDim(getNumTiles(width, PLANE_SWEEP_TILE_WIDTH), getNumTiles(height, PLANE_SWEEP_TILE_HEIGHT));
            dim3 blockDim(PLANE_SWEEP_TILE_WIDTH, PLANE_SWEEP_TILE_HEIGHT);

            planeSweepUpdateBestAndSecondBestPlaneSubPixelKernel<<<gridDim, blockDim>>>(currCosts, prev1, prev2, prevPlaneIdx, bestPlaneCosts,
                                                                                        secondBestPlaneCosts, bestPlanes, subPixelPlaneOffsets);
            PSL_CUDA_CHECK_ERROR
        }


        __global__ void planeSweepMinFloatKernel(DeviceBuffer<float> buf1, DeviceBuffer<float> buf2, DeviceBuffer<float> bufMin)
        {
            // get position of outupt
            unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
            unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

            if (x < buf1.getWidth() && y < buf1.getHeight())
            {
                const float cost1 = buf1(x,y);
                const float cost2 = buf2(x,y);

                bufMin(x,y) = min(cost1, cost2);
            }
        }

        void planeSweepMinFloat(DeviceBuffer<float>& buf1, DeviceBuffer<float>& buf2, DeviceBuffer<float>& minBuf)
        {
            dim3 gridDim(getNumTiles(buf1.getWidth(), PLANE_SWEEP_TILE_WIDTH), getNumTiles(buf1.getHeight(), PLANE_SWEEP_TILE_HEIGHT));
            dim3 blockDim(PLANE_SWEEP_TILE_WIDTH, PLANE_SWEEP_TILE_HEIGHT);

            planeSweepMinFloatKernel<<<gridDim, blockDim>>>(buf1, buf2, minBuf);
            PSL_CUDA_CHECK_ERROR
        }

        __global__ void uniqunessRatioKernel(DeviceBuffer<float> bestCost, DeviceBuffer<float> secondBestCost, DeviceBuffer<float> uniquenessRatios)
        {
            unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
            unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

            if (x < uniquenessRatios.getWidth() && y < uniquenessRatios.getHeight())
            {
                if (bestCost(x,y) == 0)
                {
                    if (secondBestCost(x,y) > 0)
                    {
                        uniquenessRatios(x,y) = 0;
                    }
                    else
                    {
                        uniquenessRatios(x,y) = 1;
                    }
                }
                else
                {
                    uniquenessRatios(x,y) = bestCost(x,y)/secondBestCost(x,y);
                }
            }
        }

        void computeUniquenessRatio(DeviceBuffer<float>& bestCost, DeviceBuffer<float>& secondBestCost, DeviceBuffer<float>& uniquenessRatios)
        {
            dim3 gridDim(getNumTiles(bestCost.getWidth(), PLANE_SWEEP_TILE_WIDTH), getNumTiles(bestCost.getHeight(), PLANE_SWEEP_TILE_HEIGHT));
            dim3 blockDim(PLANE_SWEEP_TILE_WIDTH, PLANE_SWEEP_TILE_HEIGHT);

            uniqunessRatioKernel<<<gridDim, blockDim>>>(bestCost, secondBestCost, uniquenessRatios);
            PSL_CUDA_CHECK_ERROR
        }

        __global__ void planeSweepComputeBestDepthsKernel(DeviceBuffer<int> bestPlanes, float* planesAddr, size_t planesPitch,
                                                          float* bestDepthsDAddr, size_t bestDepthsDPitch, float3 KrefInvCol1, float3 KrefInvCol2, float3 KrefInvCol3, float XiRef)
        {
            unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
            unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

            if (x < bestPlanes.getWidth() && y < bestPlanes.getHeight())
            {
                const int planeIdx = bestPlanes(x,y);

                // apply Krefinv
                float mx = KrefInvCol1.x*x + KrefInvCol3.x;
                float my = KrefInvCol2.y*y + KrefInvCol3.y;

                // apply h^-1 of camera model
                float mxPmySqr = mx*mx + my*my;
                float D = 1 + (1 - XiRef*XiRef)*mxPmySqr;

                if (D < 0)
                    D = -D;

                float fact = (XiRef + sqrtf(D))/(mxPmySqr + 1);

                float xx = fact*mx;
                float yy = fact*my;
                float zz = fact - XiRef;

                xx /= zz;
                yy /= zz;

                float3 planeN;
                planeN.x = planesAddr[planeIdx];
                planeN.y = *((float*)((char*)planesAddr + planesPitch) + planeIdx);
                planeN.z = *((float*)((char*)planesAddr + 2*planesPitch) + planeIdx);

                const float planeD = *((float*)((char*)planesAddr + 3*planesPitch) + planeIdx);

                const float denom = xx*planeN.x + yy*planeN.y + planeN.z;

                *((float*)((char*)bestDepthsDAddr + y*bestDepthsDPitch) + x) = -planeD/denom;
            }
        }


        void planeSweepComputeBestDepths(DeviceBuffer<int>& bestPlanes, int numPlanes, std::vector<float>& planes,
                                         float* bestDepthsAddr, int bestDepthsPitch, std::vector<float> &KrefInv, float XiRef)
        {
            const int width = bestPlanes.getWidth();
            const int height = bestPlanes.getHeight();

            // allocate device memory for planes
            float* planesAddr;
            size_t planesPitch;
            PSL_CUDA_CHECKED_CALL( hipMallocPitch(&planesAddr, &planesPitch, sizeof(float)*numPlanes, 4); )
            // copy planes to device
            PSL_CUDA_CHECKED_CALL( hipMemcpy2D(planesAddr, planesPitch, &(planes[0]), sizeof(float)*numPlanes, sizeof(float)*numPlanes, 4, hipMemcpyHostToDevice); )

            // allocate memory on device for best depths
            float* bestDepthsDAddr;
            size_t bestDepthsDPitch;
            PSL_CUDA_CHECKED_CALL( hipMallocPitch(&bestDepthsDAddr, &bestDepthsDPitch, sizeof(float)*width, height); )

            float3 KrefInvCol1;
            KrefInvCol1.x = KrefInv[0];
            KrefInvCol1.y = KrefInv[3];
            KrefInvCol1.z = KrefInv[6];
            float3 KrefInvCol2;
            KrefInvCol2.x = KrefInv[1];
            KrefInvCol2.y = KrefInv[4];
            KrefInvCol2.z = KrefInv[7];
            float3 KrefInvCol3;
            KrefInvCol3.x = KrefInv[2];
            KrefInvCol3.y = KrefInv[5];
            KrefInvCol3.z = KrefInv[8];

            dim3 gridDim(getNumTiles(width, PLANE_SWEEP_TILE_WIDTH), getNumTiles(height, PLANE_SWEEP_TILE_HEIGHT));
            dim3 blockDim(PLANE_SWEEP_TILE_WIDTH, PLANE_SWEEP_TILE_HEIGHT);

            planeSweepComputeBestDepthsKernel<<<gridDim, blockDim>>>(bestPlanes, planesAddr, planesPitch,
                                                                     bestDepthsDAddr, bestDepthsDPitch, KrefInvCol1, KrefInvCol2, KrefInvCol3, XiRef);
            PSL_CUDA_CHECK_ERROR

            // download result from gpu
            PSL_CUDA_CHECKED_CALL( hipMemcpy2D(bestDepthsAddr, bestDepthsPitch, bestDepthsDAddr, bestDepthsDPitch, sizeof(float)*width, height, hipMemcpyDeviceToHost); )

            // free device memory
            PSL_CUDA_CHECKED_CALL( hipFree(planesAddr); )
            PSL_CUDA_CHECKED_CALL( hipFree(bestDepthsDAddr); )

        }


        __global__ void planeSweepComputeBestDepthsSubPixelInverseKernel(DeviceBuffer<int> bestPlanes, DeviceBuffer<float> subPixelPlaneOffsets, float* planesAddr, size_t planesPitch,
                                                                  int numPlanes, float* bestDepthsDAddr, size_t bestDepthsDPitch, float3 KrefInvCol1, float3 KrefInvCol2, float3 KrefInvCol3, float XiRef)
        {
            unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
            unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

            if (x < bestPlanes.getWidth() && y < bestPlanes.getHeight())
            {
                const int planeIdx = bestPlanes(x,y);

                // apply Krefinv
                float mx = KrefInvCol1.x*x + KrefInvCol3.x;
                float my = KrefInvCol2.y*y + KrefInvCol3.y;

                // apply h^-1 of camera model
                float mxPmySqr = mx*mx + my*my;
                float D = 1 + (1 - XiRef*XiRef)*mxPmySqr;

                if (D < 0)
                    D = -D;

                float fact = (XiRef + sqrtf(D))/(mxPmySqr + 1);

                float xx = fact*mx;
                float yy = fact*my;
                float zz = fact - XiRef;

                xx /= zz;
                yy /= zz;

                float3 planeN;
                planeN.x = planesAddr[planeIdx];
                planeN.y = *((float*)((char*)planesAddr + planesPitch) + planeIdx);
                planeN.z = *((float*)((char*)planesAddr + 2*planesPitch) + planeIdx);

                float planeD = *((float*)((char*)planesAddr + 3*planesPitch) + planeIdx);

                if (subPixelPlaneOffsets(x,y) < 0)
                {
                    const int oPlaneIdx = planeIdx - 1;

                    if (oPlaneIdx >= 0)
                    {
                        float oPlaneD = *((float*)((char*)planesAddr + 3*planesPitch) + oPlaneIdx);

                        float dStep = (1.0f/oPlaneD - 1.0f/planeD)*subPixelPlaneOffsets(x,y);
                        planeD = 1.0f/(1.0f/planeD - dStep);
                    }

                }
                if (subPixelPlaneOffsets(x,y) > 0)
                {
                    const int oPlaneIdx = planeIdx + 1;

                    if (oPlaneIdx < numPlanes)
                    {
                        float oPlaneD = *((float*)((char*)planesAddr + 3*planesPitch) + oPlaneIdx);

                        float dStep = (1.0f/planeD - 1.0f/oPlaneD)*subPixelPlaneOffsets(x,y);
                        planeD = 1.0f/(1.0f/planeD - dStep);
                    }
                }

                const float denom = xx*planeN.x + yy*planeN.y + planeN.z;

                *((float*)((char*)bestDepthsDAddr + y*bestDepthsDPitch) + x) = -planeD/denom;
            }
        }


        void planeSweepComputeBestDepthsSubPixelInverse(DeviceBuffer<int>& bestPlanes, DeviceBuffer<float>& subPixelPlaneOffsets, int numPlanes, std::vector<float>& planes,
                                         float* bestDepthsAddr, int bestDepthsPitch, std::vector<float> &KrefInv, float XiRef)
        {
            const int width = bestPlanes.getWidth();
            const int height = bestPlanes.getHeight();

            // allocate device memory for planes
            float* planesAddr;
            size_t planesPitch;
            PSL_CUDA_CHECKED_CALL( hipMallocPitch(&planesAddr, &planesPitch, sizeof(float)*numPlanes, 4); )
            // copy planes to device
            PSL_CUDA_CHECKED_CALL( hipMemcpy2D(planesAddr, planesPitch, &(planes[0]), sizeof(float)*numPlanes, sizeof(float)*numPlanes, 4, hipMemcpyHostToDevice); )

            // allocate memory on device for best depths
            float* bestDepthsDAddr;
            size_t bestDepthsDPitch;
            PSL_CUDA_CHECKED_CALL( hipMallocPitch(&bestDepthsDAddr, &bestDepthsDPitch, sizeof(float)*width, height); )

            float3 KrefInvCol1;
            KrefInvCol1.x = KrefInv[0];
            KrefInvCol1.y = KrefInv[3];
            KrefInvCol1.z = KrefInv[6];
            float3 KrefInvCol2;
            KrefInvCol2.x = KrefInv[1];
            KrefInvCol2.y = KrefInv[4];
            KrefInvCol2.z = KrefInv[7];
            float3 KrefInvCol3;
            KrefInvCol3.x = KrefInv[2];
            KrefInvCol3.y = KrefInv[5];
            KrefInvCol3.z = KrefInv[8];

            dim3 gridDim(getNumTiles(width, PLANE_SWEEP_TILE_WIDTH), getNumTiles(height, PLANE_SWEEP_TILE_HEIGHT));
            dim3 blockDim(PLANE_SWEEP_TILE_WIDTH, PLANE_SWEEP_TILE_HEIGHT);

            planeSweepComputeBestDepthsSubPixelInverseKernel<<<gridDim, blockDim>>>(bestPlanes, subPixelPlaneOffsets, planesAddr, planesPitch, numPlanes,
                                                                     bestDepthsDAddr, bestDepthsDPitch, KrefInvCol1, KrefInvCol2, KrefInvCol3, XiRef);
            PSL_CUDA_CHECK_ERROR

            // download result from gpu
            PSL_CUDA_CHECKED_CALL( hipMemcpy2D(bestDepthsAddr, bestDepthsPitch, bestDepthsDAddr, bestDepthsDPitch, sizeof(float)*width, height, hipMemcpyDeviceToHost); )

            // free device memory
            PSL_CUDA_CHECKED_CALL( hipFree(planesAddr); )
            PSL_CUDA_CHECKED_CALL( hipFree(bestDepthsDAddr); )
        }


        __global__ void planeSweepComputeBestDepthsSubPixelDirectKernel(DeviceBuffer<int> bestPlanes, DeviceBuffer<float> subPixelPlaneOffsets, float* planesAddr, size_t planesPitch,
                                                                  int numPlanes, float* bestDepthsDAddr, size_t bestDepthsDPitch, float3 KrefInvCol1, float3 KrefInvCol2, float3 KrefInvCol3, float XiRef)
        {
            unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
            unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

            if (x < bestPlanes.getWidth() && y < bestPlanes.getHeight())
            {
                const int planeIdx = bestPlanes(x,y);

                // apply Krefinv
                float mx = KrefInvCol1.x*x + KrefInvCol3.x;
                float my = KrefInvCol2.y*y + KrefInvCol3.y;

                // apply h^-1 of camera model
                float mxPmySqr = mx*mx + my*my;
                float D = 1 + (1 - XiRef*XiRef)*mxPmySqr;

                if (D < 0)
                    D = -D;

                float fact = (XiRef + sqrtf(D))/(mxPmySqr + 1);

                float xx = fact*mx;
                float yy = fact*my;
                float zz = fact - XiRef;

                xx /= zz;
                yy /= zz;

                float3 planeN;
                planeN.x = planesAddr[planeIdx];
                planeN.y = *((float*)((char*)planesAddr + planesPitch) + planeIdx);
                planeN.z = *((float*)((char*)planesAddr + 2*planesPitch) + planeIdx);

                float planeD = *((float*)((char*)planesAddr + 3*planesPitch) + planeIdx);

                if (subPixelPlaneOffsets(x,y) < 0)
                {
                    const int oPlaneIdx = planeIdx - 1;

                    if (oPlaneIdx >= 0)
                    {
                        float oPlaneD = *((float*)((char*)planesAddr + 3*planesPitch) + oPlaneIdx);

                        float dStep = (planeD - oPlaneD)*subPixelPlaneOffsets(x,y);
                        planeD = planeD + dStep;
                    }

                }
                if (subPixelPlaneOffsets(x,y) > 0)
                {
                    const int oPlaneIdx = planeIdx + 1;


                    if (oPlaneIdx < numPlanes)
                    {
                        float oPlaneD = *((float*)((char*)planesAddr + 3*planesPitch) + oPlaneIdx);

                        float dStep = (oPlaneD - planeD)*subPixelPlaneOffsets(x,y);
                        planeD = planeD + dStep;
                    }
                }

                const float denom = xx*planeN.x + yy*planeN.y + planeN.z;

                *((float*)((char*)bestDepthsDAddr + y*bestDepthsDPitch) + x) = -planeD/denom;
            }
        }


        void planeSweepComputeBestDepthsSubPixelDirect(DeviceBuffer<int>& bestPlanes, DeviceBuffer<float>& subPixelPlaneOffsets, int numPlanes, std::vector<float>& planes,
                                                       float* bestDepthsAddr, int bestDepthsPitch, std::vector<float> &KrefInv, float XiRef)
        {
            const int width = bestPlanes.getWidth();
            const int height = bestPlanes.getHeight();

            // allocate device memory for planes
            float* planesAddr;
            size_t planesPitch;
            PSL_CUDA_CHECKED_CALL( hipMallocPitch(&planesAddr, &planesPitch, sizeof(float)*numPlanes, 4); )
            // copy planes to device
            PSL_CUDA_CHECKED_CALL( hipMemcpy2D(planesAddr, planesPitch, &(planes[0]), sizeof(float)*numPlanes, sizeof(float)*numPlanes, 4, hipMemcpyHostToDevice); )

            // allocate memory on device for best depths
            float* bestDepthsDAddr;
            size_t bestDepthsDPitch;
            PSL_CUDA_CHECKED_CALL( hipMallocPitch(&bestDepthsDAddr, &bestDepthsDPitch, sizeof(float)*width, height); )

            float3 KrefInvCol1;
            KrefInvCol1.x = KrefInv[0];
            KrefInvCol1.y = KrefInv[3];
            KrefInvCol1.z = KrefInv[6];
            float3 KrefInvCol2;
            KrefInvCol2.x = KrefInv[1];
            KrefInvCol2.y = KrefInv[4];
            KrefInvCol2.z = KrefInv[7];
            float3 KrefInvCol3;
            KrefInvCol3.x = KrefInv[2];
            KrefInvCol3.y = KrefInv[5];
            KrefInvCol3.z = KrefInv[8];

            dim3 gridDim(getNumTiles(width, PLANE_SWEEP_TILE_WIDTH), getNumTiles(height, PLANE_SWEEP_TILE_HEIGHT));
            dim3 blockDim(PLANE_SWEEP_TILE_WIDTH, PLANE_SWEEP_TILE_HEIGHT);

            planeSweepComputeBestDepthsSubPixelDirectKernel<<<gridDim, blockDim>>>(bestPlanes, subPixelPlaneOffsets, planesAddr, planesPitch, numPlanes,
                                                                     bestDepthsDAddr, bestDepthsDPitch, KrefInvCol1, KrefInvCol2, KrefInvCol3, XiRef);
            PSL_CUDA_CHECK_ERROR

            // download result from gpu
            PSL_CUDA_CHECKED_CALL( hipMemcpy2D(bestDepthsAddr, bestDepthsPitch, bestDepthsDAddr, bestDepthsDPitch, sizeof(float)*width, height, hipMemcpyDeviceToHost); )

            // free device memory
            PSL_CUDA_CHECKED_CALL( hipFree(planesAddr); )
            PSL_CUDA_CHECKED_CALL( hipFree(bestDepthsDAddr); )

        }

        __forceinline__ __device__ float computeWarpedGrayscaleTexturePixel(int x , int y, int width, int height,
                                                                            float krefinv11, float krefinv13, float krefinv22, float krefinv23, float xiref,
                                                                            float kother11, float kother13, float kother22, float kother23, float xiother,
                                                                            float rt11, float rt12, float rt13, float rt21, float rt22, float rt23, float rt31, float rt32, float rt33)
        {
            //apply Krefinv
            const float mx = krefinv11*x + krefinv13;
            const float my = krefinv22*y + krefinv23;

            // apply h^-1 of camera model
            const float mxPmySqr = mx*mx + my*my;

            const float fact = (xiref + sqrtf(abs(1 + (1 - xiref*xiref)*mxPmySqr)))/(mxPmySqr + 1);

            const float xx = fact*mx;
            const float yy = fact*my;
            const float zz = fact - xiref;

            // apply rt
            const float xxw = rt11*xx + rt12*yy + rt13*zz;
            const float yyw = rt21*xx + rt22*yy + rt23*zz;
            const float zzw = rt31*xx + rt32*yy + rt33*zz;

            // apply h
            const float lengthInv = 1.0f/sqrtf(xxw*xxw + yyw*yyw + zzw*zzw);
            const float zzzwInv = 1.0f/(zzw*lengthInv + xiother);

            // apply Kother

            const float xw = kother11*xxw*lengthInv*zzzwInv + kother13;
            const float yw = kother22*yyw*lengthInv*zzzwInv + kother23;

            const float u = (xw+0.5f) / (float) width;
            const float v = (yw+0.5f) / (float) height;

            const float1 pix = tex2D(planeSweepGrayscaleTexture, u, v);

            return pix.x*255;
        }

        __forceinline__ __device__ float computeZNCC(float normalizer, float ref, float refSqr, float other, float otherSqr, float prod)
        {

            const float numerator = prod - normalizer*(ref*other);

            float denomRef = refSqr - normalizer*ref*ref;
            float denomOther = otherSqr - normalizer*other*other;

            if (denomRef < 2)
                denomRef = 2;

            if (denomOther < 2)
                denomOther = 2;

            return (1 - (numerator/(sqrtf(denomRef)*sqrtf(denomOther))))/2.0f;
        }
    }
}
